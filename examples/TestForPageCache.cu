#include <cassert>
#include <iostream>

#include "geminifs_api.cuh"

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer() {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer() {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start() {
            hipEventRecord(start, 0);
      }

      void Stop() {
            hipEventRecord(stop, 0);
      }

      float Elapsed() {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int
main() {
  int nr_warps = 16;
  size_t nr_pages = nr_warps * 32;
  size_t page_size = 128 * (1ull << 10)/*KB*/;
  size_t page_capacity = nr_pages * page_size;
  size_t virtual_space_size = page_capacity * 64;

  uint64_t *dev_buf1;
  uint64_t *dev_buf2;

  gpuErrchk(hipMallocManaged(&dev_buf1, virtual_space_size));
  gpuErrchk(hipMallocManaged(&dev_buf2, virtual_space_size));

  dev_fd_t dev_fd = host_get_pagecache__for_test_evicting(virtual_space_size, page_capacity, page_size, nr_warps);

  //device_xfer_geminifs_file<<<nr_warps, 32>>>(dev_fd, 0, dev_buf1, virtual_space_size, 0);
  //hipDeviceSynchronize();
  device_xfer_geminifs_file<<<nr_warps, 32>>>(dev_fd, 0, dev_buf2, virtual_space_size, 1);
  hipDeviceSynchronize();

  return 0;
}

