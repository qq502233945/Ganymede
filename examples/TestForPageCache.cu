#include <cassert>
#include <iostream>

#include "geminifs_api.cuh"

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer() {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer() {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start() {
            hipEventRecord(start, 0);
      }

      void Stop() {
            hipEventRecord(stop, 0);
      }

      float Elapsed() {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int
main() {
  size_t nr_pages = 108;
  size_t dev_page_size = 128 * (1ull << 10)/*KB*/;
  size_t page_capacity = nr_pages * dev_page_size;
  size_t virtual_space_size = page_capacity * 108;

  uint64_t *dev_buf1;
  uint64_t *dev_buf2;

  gpuErrchk(hipMallocManaged(&dev_buf1, virtual_space_size));
  gpuErrchk(hipMallocManaged(&dev_buf2, virtual_space_size));

  dev_fd_t dev_fd = host_get_pagecache__for_test_evicting(virtual_space_size, page_capacity, dev_page_size);

  //device_xfer_geminifs_file<<<2, 32>>>(dev_fd, 0, dev_buf1, virtual_space_size, 0);
  //hipDeviceSynchronize();
  device_xfer_geminifs_file<<<108, 32>>>(dev_fd, 0, dev_buf2, virtual_space_size, 1);
  hipDeviceSynchronize();

  return 0;
}

