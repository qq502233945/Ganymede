#include <cassert>
#include <iostream>

#include "geminifs_api.cuh"

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer() {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer() {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start() {
            hipEventRecord(start, 0);
      }

      void Stop() {
            hipEventRecord(stop, 0);
      }

      float Elapsed() {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int
main() {
  size_t virtual_space_size = 256 * (1ull << 20)/*GB*/;
  size_t page_capacity = 128 * (1ull << 20);
  size_t dev_page_size = 4096;

  uint64_t *dev_buf1;
  uint64_t *dev_buf2;

  gpuErrchk(hipMallocManaged(&dev_buf1, virtual_space_size));
  gpuErrchk(hipMallocManaged(&dev_buf2, virtual_space_size));

  dev_fd_t dev_fd = host_get_pagecache__for_test_evicting(virtual_space_size, page_capacity, dev_page_size);

  device_xfer_geminifs_file<<<108, 32>>>(dev_fd, 0, dev_buf1, virtual_space_size, 0);
  hipDeviceSynchronize();
  device_xfer_geminifs_file<<<108, 32>>>(dev_fd, 0, dev_buf2, virtual_space_size, 1);
  hipDeviceSynchronize();

  return 0;
}

