#include "hip/hip_runtime.h"
#include "geminifs_api.cuh"
#include "geminifs_internal.cuh"

#include <ctrl.h>
#include <buffer.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_types.h>
#include <nvm_util.h>
#include <nvm_cmd.h>
#include "get-offset/get-offset.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

class QueueAcquireHelper {
public:
    __device__
    QueueAcquireHelper(int nr_queues) {
        this->nr_queues = nr_queues;
        this->locks = new cuda::binary_semaphore<cuda::thread_scope_device> [nr_queues];
        for (size_t i = 0; i < nr_queues; i++)
            this->locks[i].release();
    }

    __forceinline__ __device__ int
    acquire_queue() {
        //int queue = get_smid() % this->nr_queues;
        int queue = 0;
        this->locks[queue].acquire();
        return queue;
    }

    __forceinline__ __device__ void
    release_queue(int queue) {
        this->locks[queue].release();
    }

private:
    int nr_queues;
    cuda::binary_semaphore<cuda::thread_scope_device> *locks;
};

class CachePage_NvmeBacking: public CachePage {
public:
    __device__
    CachePage_NvmeBacking(int page_size): CachePage(page_size, nullptr, 0) { }

    DmaPtr gpu_buffer;

    int n_ioaddrs;
    uint64_t *ioaddrs;
    int hqps_block_size_log;
private:
    __device__ __forceinline__ nvme_ofst_t
    __get_nvmeofst(struct geminiFS_hdr *hdr, vaddr_t va) {
        uint64_t l1_idx = va >> hdr->block_bit;
        if (l1_idx < hdr->nr_l1)
            return hdr->l1[l1_idx];
        return 0;
    }

    __device__ void
    __write_back(FilePageId filepage_id, void *ctrl, void *hdr, void *queue_acquire_helper) {
        this->__xfer(filepage_id, ctrl, hdr, queue_acquire_helper, 0);
    }

    __device__ void
    __read_in(FilePageId filepage_id, void *ctrl, void *hdr, void *queue_acquire_helper) {
        this->__xfer(filepage_id, ctrl, hdr, queue_acquire_helper, 1);
    }

    __device__ __forceinline__ void
    __xfer(FilePageId filepage_id, void *ctrl_, void *hdr_, void *queue_acquire_helper_, int is_read) {
        auto *ctrl = (Controller *)ctrl_;
        auto *hdr = (struct geminiFS_hdr *)hdr_;
        auto *queue_acquire_helper = (QueueAcquireHelper *)queue_acquire_helper_;


        int page_size = this->page_size;
        int page_bit = __popc(page_size - 1);
        int file_block_size = 1 << hdr->block_bit;



        assert(file_block_size == page_size);

        vaddr_t file_va = filepage_id << page_bit;
        nvme_ofst_t nvme_ofst = this->__get_nvmeofst(hdr, file_va);

        int queue = queue_acquire_helper->acquire_queue();
        printf("I get a queue [%llx]\n", (uint64_t)queue);
        QueuePair* qp = &ctrl->d_qps[queue];

        size_t start_hqps_block = nvme_ofst >> this->hqps_block_size_log;
        int nr_hqps_blocks = file_block_size >> this->hqps_block_size_log;

        int size_of_ioaddr = 4096;
        int nr_hqps_blocks__per_ioaddr = 4096 / (1 << this->hqps_block_size_log);

        for (size_t idx_ioaddr = 0;
                idx_ioaddr < page_size / size_of_ioaddr;
                idx_ioaddr++, start_hqps_block += nr_hqps_blocks__per_ioaddr) {
            nvm_cmd_t cmd;
            uint16_t cid = get_cid(&(qp->sq));
            uint64_t prp1 = this->ioaddrs[idx_ioaddr];
            uint64_t prp2 = 0;
            {
                if (is_read) {
                    nvm_cmd_header(&cmd, cid, NVM_IO_READ, qp->nvmNamespace);
                    printf("read in filepage_id[%llx] file_va[%llx] nvmeofst[%llx] ioaddr[%llx] hqps_block_size_log[%llx]\n", filepage_id, file_va, nvme_ofst, this->ioaddrs[idx_ioaddr], (uint64_t)this->hqps_block_size_log);
                } else {
                    nvm_cmd_header(&cmd, cid, NVM_IO_WRITE, qp->nvmNamespace);
                    printf("write back filepage_id[%llx] file_va[%llx] nvmeofst[%llx] ioaddr[%llx] hqps_block_size_log[%llx]\n", filepage_id, file_va, nvme_ofst, this->ioaddrs[idx_ioaddr], (uint64_t)this->hqps_block_size_log);
                }

                nvm_cmd_data_ptr(&cmd, prp1, prp2);
                nvm_cmd_rw_blks(&cmd, start_hqps_block, nr_hqps_blocks__per_ioaddr);
                uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);
                uint32_t head, head_;
                uint32_t cq_pos = cq_poll(&qp->cq, cid, &head, &head_);
                qp->cq.tail.fetch_add(1, simt::memory_order_acq_rel);
                cq_dequeue(&qp->cq, cq_pos, &qp->sq, head, head_);
                put_cid(&qp->sq, cid);
            }
            if (!is_read) {
                    nvm_cmd_header(&cmd, cid, NVM_IO_FLUSH, qp->nvmNamespace);
                nvm_cmd_data_ptr(&cmd, prp1, prp2);
                nvm_cmd_rw_blks(&cmd, start_hqps_block, nr_hqps_blocks__per_ioaddr);
                uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);
                uint32_t head, head_;
                uint32_t cq_pos = cq_poll(&qp->cq, cid, &head, &head_);
                qp->cq.tail.fetch_add(1, simt::memory_order_acq_rel);
                cq_dequeue(&qp->cq, cq_pos, &qp->sq, head, head_);
                put_cid(&qp->sq, cid);
            }
        }

        printf("I release the queue [%llx]\n", (uint64_t)queue);
        queue_acquire_helper->release_queue(queue);
    }
};

static Controller *ctrl;

void
host_open_all(
        const char *snvme_control_path,
        const char *snvme_path,
        const char *nvme_dev_path,
        const char *mount_path,
        uint32_t ns_id,
        uint64_t queueDepth,
        uint64_t numQueues) {
    int device;
    gpuErrchk(hipGetDevice(&device));
    ctrl = new Controller(
            (char *)snvme_control_path,
            (char *)snvme_path,
            (char *)nvme_dev_path,
            (char *)mount_path,
            ns_id, device, queueDepth, numQueues);
}

void
host_close_all() {
    delete ctrl;
}

dev_fd_t
host_open_geminifs_file_for_device(
        host_fd_t host_fd,
        uint64_t pagecache_capacity,
        int page_size) {
    struct geminiFS_hdr *hdr = host_fd;


    int file_block_size = 1 << hdr->block_bit;
    if (file_block_size < (128 * (1ull << 10))) {
        assert(file_block_size == page_size);
    } else {
        assert(file_block_size <= page_size);
    }

    struct geminiFS_hdr *hdr__dev;
    gpuErrchk(hipMallocManaged(&hdr__dev, hdr->first_block_base));
    assert((off_t)(-1) != lseek(hdr->fd, 0, SEEK_SET));
    assert(hdr->first_block_base == read(hdr->fd, hdr__dev, hdr->first_block_base));

    size_t nr_page = pagecache_capacity / page_size;

    CachePage_NvmeBacking *cachepage_structures;
    gpuErrchk(hipMallocManaged(&cachepage_structures, sizeof(CachePage_NvmeBacking) * nr_page));

    CachePage **pages;
    gpuErrchk(hipMalloc(&pages, sizeof(CachePage *) * nr_page));

    int nr_queues = ctrl->n_qps;
    QueueAcquireHelper *queue_acquire_helper;
    gpuErrchk(hipMalloc(&queue_acquire_helper, sizeof(QueueAcquireHelper)));

    RUN_ON_DEVICE({
        for (size_t i = 0; i < nr_page; i++) {
            auto *cachepage = cachepage_structures + i;
            pages[i] = new (cachepage) CachePage_NvmeBacking (page_size);
        }
        new (queue_acquire_helper) QueueAcquireHelper (nr_queues);
    });

    int device;
    gpuErrchk(hipGetDevice(&device));

    for (size_t i = 0; i < nr_page; i++) {
        auto *cachepage = cachepage_structures + i;
        cachepage->gpu_buffer = createDma(ctrl->ctrl, page_size, device);
        cachepage->buf = cachepage->gpu_buffer->vaddr;
        cachepage->n_ioaddrs = cachepage->gpu_buffer->n_ioaddrs;
        gpuErrchk(hipMallocManaged(&(cachepage->ioaddrs),
                    sizeof(uint64_t) * cachepage->gpu_buffer->n_ioaddrs));
        for (size_t j = 0; j < cachepage->gpu_buffer->n_ioaddrs; j++)
            cachepage->ioaddrs[j] = cachepage->gpu_buffer->ioaddrs[j];
        cachepage->hqps_block_size_log = ctrl->h_qps[0]->block_size_log;
    }


    return __internal__get_pagecache(pagecache_capacity,
            page_size,
            hdr->virtual_space_size,
            pages,
            nr_queues,
            ctrl->d_ctrl_ptr, hdr__dev, queue_acquire_helper);
}

class CachePage_TestForPageCache: public CachePage {
public:
    __device__
    CachePage_TestForPageCache(int page_size): CachePage(page_size, nullptr, 0) { }
private:
    __device__ __forceinline__ nvme_ofst_t
    __get_nvmeofst(struct geminiFS_hdr *hdr, vaddr_t va) {
        return 0;
    }

    __device__ void
    __write_back(FilePageId filepage_id, void *ctrl, void *hdr, void *queue_acquire_helper) {
        this->__xfer(filepage_id, ctrl, hdr, queue_acquire_helper, 1);
    }

    __device__ void
    __read_in(FilePageId filepage_id, void *ctrl, void *hdr, void *queue_acquire_helper) {
        this->__xfer(filepage_id, ctrl, hdr, queue_acquire_helper, 0);
    }

    __device__ __forceinline__ void
    __xfer(FilePageId filepage_id, void *ctrl_, void *hdr_, void *queue_acquire_helper_, int is_read) {
        auto *queue_acquire_helper = (QueueAcquireHelper *)queue_acquire_helper_;


        int queue = queue_acquire_helper->acquire_queue();
        //__nanosleep(1000);
        queue_acquire_helper->release_queue(queue);

    }
};

dev_fd_t
host_get_pagecache__for_test_evicting(
        uint64_t fake_file_size,
        uint64_t pagecache_capacity,
        int page_size) {
    size_t nr_page = pagecache_capacity / page_size;

    uint8_t *all_raw_pages;
    gpuErrchk(hipMalloc(&all_raw_pages, nr_page * page_size));

    CachePage_TestForPageCache *cachepage_structures;
    gpuErrchk(hipMallocManaged(&cachepage_structures, sizeof(CachePage_TestForPageCache) * nr_page));

    CachePage **pages;
    gpuErrchk(hipMalloc(&pages, sizeof(CachePage *) * nr_page));

    int nr_queues = 32;
    QueueAcquireHelper *queue_acquire_helper;
    gpuErrchk(hipMalloc(&queue_acquire_helper, sizeof(QueueAcquireHelper)));

    RUN_ON_DEVICE({
        for (size_t i = 0; i < nr_page; i++) {
            auto *cachepage = cachepage_structures + i;
            pages[i] = new (cachepage) CachePage_TestForPageCache (page_size);
            cachepage->buf = all_raw_pages + i * page_size;
        }
        new (queue_acquire_helper) QueueAcquireHelper (nr_queues);
    });

    return __internal__get_pagecache(pagecache_capacity,
            page_size,
            fake_file_size,
            pages,
            nr_queues,
            nullptr, nullptr, queue_acquire_helper);
}
